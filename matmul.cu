#include <stdint.h> // Required for uint32_t
#include <stdio.h>
#include <stdlib.h> // For rand()
#include <sys/sysinfo.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>

struct timespec begin, end;
double elapsed;
#define MAT_N 800
#define MAT_M 800
float *mat_A;//[MAT_M*MAT_N];
float *mat_B;//[MAT_N*MAT_M];
float *mat_C_cpu;//[MAT_M*MAT_M]; // results in MxM matrix
float *mat_C_gpu;//[MAT_M*MAT_M]; // results in MxM matrix

/*
void printMatrix(float* matrix, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%.2f ", matrix[i * cols + j]);
        }
        printf("\n");
    }
}
*/

__host__ __device__ inline void matMul_base(int i, float* A, float* B, float* C){
  float current;
  for (int j = 0; j < MAT_M; j++) {
    current = 0;
    for (int k = 0; k < MAT_N; k++) {
      current += A[i*MAT_M + k] * B[k*MAT_N + j];
    }
    C[i*MAT_M + j] = current;
  }
}

void matMul_CPU(){
  for (int i = 0; i < MAT_M; i++) {
    matMul_base(i, mat_A, mat_B, mat_C_cpu);
  }
}

__global__ void matMul_GPU(float* mat_A, float* mat_B, float* mat_C_gpu){
  int global_thread_id = blockIdx.x*blockDim.x+threadIdx.x;
  matMul_base(global_thread_id, mat_A, mat_B, mat_C_gpu);
}

int main(int argc, char *argv[]) {
  hipMallocManaged(&mat_A, MAT_M*MAT_N*sizeof(float));
  hipMallocManaged(&mat_B, MAT_N*MAT_M*sizeof(float));
  hipMallocManaged(&mat_C_cpu, MAT_M*MAT_M*sizeof(float));
  hipMallocManaged(&mat_C_gpu, MAT_M*MAT_M*sizeof(float));

  clock_gettime(CLOCK_MONOTONIC, &begin);

  printf("This system has %d processors configured and "
         "%d processors available.\n",
         get_nprocs_conf(), get_nprocs());

  printf("initializing\n");

  for (int i = 0; i < MAT_M; i++) {
    for (int j = 0; j < MAT_N; j++) {
      mat_A[i*MAT_M + j] = i+j;
      mat_B[j*MAT_N + i] = i+j;
    }
  }

  printf("done initializing\n");

  matMul_CPU();
  clock_gettime(CLOCK_MONOTONIC, &end);

  elapsed = end.tv_sec - begin.tv_sec;
  elapsed += (end.tv_nsec - begin.tv_nsec) / 1000000000.0;
  printf("took %f s\n", elapsed);


  ////
  ////
  ////
  ////
  ////

  clock_gettime(CLOCK_MONOTONIC, &begin);
  int num_blocks = (MAT_M + 32 - 1)/32;
  printf("num blocks is: %d\n", num_blocks);
  matMul_GPU<<<num_blocks,32>>>(mat_A, mat_B, mat_C_gpu);

  hipError_t err = hipSuccess;
  if (hipGetLastError() != hipSuccess){
      fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
  }

  hipDeviceSynchronize();
  clock_gettime(CLOCK_MONOTONIC, &end);

  elapsed = end.tv_sec - begin.tv_sec;
  elapsed += (end.tv_nsec - begin.tv_nsec) / 1000000000.0;
  printf("took %f s\n", elapsed);

  for (int i = 0; i < MAT_M * MAT_M; i++){
     if (fabs(mat_C_gpu[i] - mat_C_cpu[i]) > 1e-5){
       printf("no honey\n");
     }
  }
  printf("yes honey\n");

  return 0;
}
